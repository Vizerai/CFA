#include "CFA.cuh"
#include "CFA.h"

#define DEBUG		1

#if(DEBUG == 1)
#define MV_MULTIPLY(A, B, C)	cusp::multiply(A, B, C);\
								fprintf(stdout, "(%dx%d) * (%dx1) -> (%dx1)\n", A.num_rows, A.num_cols, B.size(), C.size())

#define DEBUG_PRINT(A, B)		fprintf(stdout, A);\
								cusp::print(B);
#else
#define MULTIPLY(A, B, C)		cusp::multiply(A, B, C)
#define DEBUG_PRINT(A, B)
#endif

#include "host_forms.inl"
#include "device_forms.inl"

// template <typename INDEX_TYPE, typename MEM_TYPE>
// inline void subtract_vec(cusp::array1d<INDEX_TYPE, MEM_TYPE> &A,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &B,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &C)
// {
// 	thrust::transform(A.begin(), A.end(), B.begin(), C.begin(), thrust::minus<INDEX_TYPE>());
// }

// General GPU Device CUDA Initialization
template <typename INDEX_TYPE, typename MEM_TYPE>
int CFA<INDEX_TYPE, MEM_TYPE>::gpuDeviceInit(int devID)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stdout, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stdout, "\n");
        fprintf(stdout, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stdout, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stdout, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

    if (deviceProp.major < 1)
    {
        fprintf(stdout, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);
    }
    
    checkCudaErrors( hipSetDevice(devID) );
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::Init()
{
	size_t vec_size = sigma.num_rows;
	VOID_vec.resize(vec_size, 0);
	NOT_FALSE_vec.resize(vec_size, 1);
	FALSE_vec.resize(vec_size, 0);
	BOOL_vec.resize(vec_size, 0);
	NUM_vec.resize(vec_size, 0); 
	LIST_vec.resize(vec_size, 0);
	tb.resize(1, 0);
	fb.resize(1, 0);
	AND_vec.resize(r.size(), 0);

	for(int i=0; i<ARG_MAX; ++i)
	{
		v[i].resize(vec_size, 0);
		a[i].resize(sigma.num_cols, 0);
		
		if(Call[i].size() != r.size())
			Call[i].resize(r.size(), 0);
		if(PrimList[i].size() != r.size())
			PrimList[i].resize(r.size(), 0);
	}
    a_var.resize(sigma.num_cols, 0);
    a_set.resize(sigma.num_cols, 0);
    vf.resize(vec_size, 0);
    v_set.resize(vec_size, 0);
    v_cond.resize(vec_size, 0);
    v_list.resize(vec_size, 0);

	for(int i=0; i<vec_size; ++i)
	{
		if(i == vec_size - 5)			//list
			LIST_vec[i] = 1;			
		else if(i == vec_size - 4)		//void
			VOID_vec[i] = 1;
		else if(i == vec_size - 3)		//#t
			BOOL_vec[i] = 1;
		else if(i == vec_size - 2)		//#f
		{
			NOT_FALSE_vec[i] = 0;
			BOOL_vec[i] = 1;
			FALSE_vec[i] = 1;
		}
		else if(i == vec_size - 1)		//NUM
			NUM_vec[i] = 1;
	}

	#if(BUILD_TYPE == GPU)
	temp_r.resize(r.size());
	temp_vec.resize(sigma.num_cols);
	temp_Body.resize(Body.num_rows, Body.num_rows, Body.num_entries, Body.num_rows/4);
	//vf_mat.resize(Body.num_rows, r.size(), Body.num_entries, Body.num_entries/4);
	temp_Fun = Fun;
	temp_Arg = Arg[0];
	print_matrix_info(temp_Mat[0]);
	temp_Mat[0].resize(sigma.num_rows, sigma.num_cols, 0, sigma.num_cols/8, 32);
	temp_Mat[1].resize(sigma.num_rows, sigma.num_cols, 0, sigma.num_cols/8, 32);
	temp_Mat[2].resize(sigma.num_rows, sigma.num_cols, 0, sigma.num_cols/8, 32);
	print_matrix_info(temp_Mat[0]);
	a_var_mat = Fun;
	for(int i=0; i<m_maxCall; ++i)
		v_mat[i].resize(Body.num_rows, r.size(), Body.num_entries, Body.num_entries/4);
	#endif

	GRID = dim3(1,1,1);
	BLOCK = dim3(BLOCK_SIZE,1,1);
}

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::ReadTestFile(const char* filename)
{
	std::ifstream tf(filename);
	if(tf.fail())
	{
		fprintf(stdout, "Invalid test file: %s\n", filename);
		exit(1);
	}

	fprintf(stdout, "Reading test file: %s\n", filename);
	while(!tf.eof())
	{
		int rows, cols, i, j;
		char buf[64], name[32];

		tf.getline(buf, 64);
		if(tf.gcount() > 1)
		{
			std::sscanf(buf, "%s %d %d", name, &rows, &cols);

			std::vector< std::pair<int, int> > indices;
			tf.getline(buf, 64);
			while(tf.gcount() > 1)
			{
				std::sscanf(buf, "%d %d", &i, &j);
				if(i<0 || i>=rows || j<0 || j>=cols)
					fprintf(stdout, "ERROR Rows: %d Cols: %d \t (i:%d j:%d)\n", rows, cols, i, j);

				indices.push_back(std::make_pair(i,j));
				tf.getline(buf, 64);
			}
			std::sort(indices.begin(), indices.end(), compare_entry);

			#define NUM_ENTRIES_PER_ROW		32
			cusp::coo_matrix<int, int, cusp::host_memory> A(rows, cols, indices.size()), B;
			for(int i=0; i<A.num_entries; ++i)
			{
				A.row_indices[i] = indices[i].first;
				A.column_indices[i] = indices[i].second;
				A.values[i] = 1;
			}
			//A.sort_by_row_and_column();

			cusp::array1d<int, cusp::host_memory> vec;
			if(A.num_cols == 1)
			{
				vec.resize(A.num_rows, 0);
				for(int i=0; i<A.num_entries; ++i)
					vec[A.row_indices[i]] = 1;
			}

			fprintf(stderr, "\n%s (%d x %d)\n", name, A.num_rows, A.num_cols);
			B.resize(A.num_rows, A.num_cols, A.num_entries);
			cusp::transpose(A, B);
			fprintf(stderr, "B: (%d x %d)\n", B.num_rows, B.num_cols);
			cusp::print(B);

			std::string mat_name(name), sname = "", snum = "";
			int mat_num = -1;
			for(int i=0; i<NUM_MATRIX_TYPES; ++i)
			{
				std::string find_name = MatrixTypeMap[i];
				if(mat_name.find(find_name) == 0)
				{
					sname = find_name;
					snum = mat_name.substr(sname.size());
					if(snum.size() > 0)
						mat_num = atoi(snum.c_str());
					break;
				}
			}

			//parse name
			if(sname == "r")
				r = vec;
			else if(sname == "sigma")
				sigma = B;
			else if(sname == "CondTrue")
				CondTrue = B;
			else if(sname == "CondFalse")
				CondFalse = B;
			else if(sname == "Body")
				Body = B;
			else if(sname == "Fun")
				Fun = B;
			else if(sname == "Arg")
				Arg[mat_num-1] = B;
			else if(sname == "Var")
				Var[mat_num-1] = B;
			else if(sname == "Call")
			{
				Call[mat_num] = vec;
				if(m_maxCall < mat_num)
					m_maxCall = mat_num;
			}
			else if(sname == "PrimBool")
				PrimBool = vec;
			else if(sname == "PrimNum")
				PrimNum = vec;
			else if(sname == "PrimVoid")
				PrimVoid = vec;
			else if(sname == "PrimList")
			{
				PrimList[mat_num] = vec;
				if(m_maxList < mat_num)
					m_maxList = mat_num;
			}
			else if(sname == "If")
				IF = vec;
			else if(sname == "Set")
				SET = vec;
			else
				fprintf(stdout, "could not match input matrix: %s\n", name);
		}
	}

	tf.close();
}

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::Run_Analysis()
{
	fprintf(stdout, "\n\n\nStarting analysis...\n");

	s.resize(r.size(), 0);
	r_prime = r;
	sigma_prime = sigma;
	int iter=0;
	bool sigma_change = false, r_change = false;
	fprintf(stderr, "m_maxCall: %d  m_maxList: %d\n", m_maxCall, m_maxList);
	cusp::ell_matrix<int, INDEX_TYPE, cusp::host_memory> temp_sigma;
	//cusp::print(sigma);
	//typedef r.memory_space MemorySpace;

	debug = false;
	do
	{
		//CPU execution
		fprintf(stdout, "\n\nITERATION %d\n\n", ++iter);

		//f_call
		f_call();

		//f_list
		f_list();

		//f_set
		f_set();

		//f_if
		f_if();

		//f_primBool
		f_primBool();

		//f_primNum
		f_primNum();

		//f_primVoid
		f_primVoid();

		//correct values...  (this won't be a problem in later versions)
		// for(int i=0; i<sigma_prime.num_entries; ++i)
		// 	sigma_prime.values[i] = 1;
		for(int i=0; i<r_prime.size(); ++i)
			if(r_prime[i])
				r_prime[i] = 1;

		fprintf(stdout, "\nupdate sigma\n");
		// cusp::subtract(sigma_prime, sigma, temp_Mat[0]);
		if(debug)
		{
		 	fprintf(stdout, "***sigma difference***\n");
			//cusp::print(temp_Mat[0]);

			temp_sigma = sigma_prime;
			int num_entries = 0;
			int pitch = temp_sigma.column_indices.pitch;
			const INDEX_TYPE invalid_index = cusp::ell_matrix<int, INDEX_TYPE, cusp::device_memory>::invalid_index;
			for(int col=0; col<temp_sigma.column_indices.num_cols; ++col)
			{	
				int offset = pitch*col;
				for(int row=0; row<temp_sigma.num_rows; ++row, ++offset)
				{
					if(temp_sigma.column_indices.values[offset] != invalid_index)
						num_entries++;
				}
			}
			temp_sigma.num_entries = num_entries;
			cusp::print(temp_sigma);
		}
		if(sigma.num_entries != sigma_prime.num_entries)
			sigma_change = true;
		else
			sigma_change = false;

		fprintf(stdout, "\nupdate r\n");
		if(r.size() == temp_r.size())
			thrust::transform(r.begin(), r.end(), r_prime.begin(), temp_r.begin(), thrust::minus<INDEX_TYPE>());
		// if(debug)
		// {
		// 	fprintf(stdout, "***r difference***\n");
		// 	cusp::print(temp_r);
		// 	cusp::print(r_prime);
		// }
		r_change = false;
		for(int i=0; i<temp_r.size(); ++i)
			if(temp_r[i] != 0)
			{
				r_change = true;
				break;
			}

		r = r_prime;
		sigma = sigma_prime;
		fprintf(stdout, "end iteration\n");

	} while(r_change || sigma_change);

	fprintf(stdout, "Analysis Complete...\n");
}

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::WriteStore()
{
	// fprintf(stdout, "\nr:\n");
	// cusp::print(r);

	cusp::coo_matrix<int, INDEX_TYPE, MEM_TYPE> store;
	cusp::transpose(sigma, store);
	// fprintf(stdout, "\nsigma:\n");
	// cusp::print(store);

	std::ofstream output("tests/output.dat");
	output << "sigma " << store.num_rows << " " << store.num_cols << std::endl;
	for(int i=0; i<store.num_entries; ++i)
	{
		output << store.row_indices[i] << " " << store.column_indices[i] << " " << std::endl;
	}

	output.close();
}

void CPU_Test(std::string filename)
{
#if(BUILD_TYPE == CPU)
	CFA<int, cusp::host_memory> Analysis;
#elif(BUILD_TYPE == GPU)
	CFA<int, cusp::device_memory> Analysis;
	Analysis.gpuDeviceInit(1);
#endif	

	Analysis.ReadTestFile(filename.c_str());
	Analysis.Init();

	double startTime = omp_get_wtime();
	Analysis.Run_Analysis();
	double endTime = omp_get_wtime();

	fprintf(stdout, "Run Time: %f seconds\n", endTime - startTime);
	Analysis.WriteStore();
}

void GPU_Test()
{

}