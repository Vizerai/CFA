#include "hip/hip_runtime.h"
#include "CFA.cuh"
#include "CFA.h"

#define DEBUG		1

#if(DEBUG == 1)
#define MV_MULTIPLY(A, B, C)	cusp::multiply(A, B, C);\
								fprintf(stdout, "(%dx%d) * (%dx1) -> (%dx1)\n", A.num_rows, A.num_cols, B.size(), C.size())

#define DEBUG_PRINT(A, B)		fprintf(stdout, A);\
								cusp::print(B);
#else
#define MULTIPLY(A, B, C)		cusp::multiply(A, B, C)
#define DEBUG_PRINT(A, B)
#endif

#include "host_forms.inl"
#include "device_forms.inl"

// template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
// inline void subtract_vec(cusp::array1d<INDEX_TYPE, MEM_TYPE> &A,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &B,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &C)
// {
// 	thrust::transform(A.begin(), A.end(), B.begin(), C.begin(), thrust::minus<INDEX_TYPE>());
// }

// General GPU Device CUDA Initialization
template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
int CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::gpuDeviceInit(int devID)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stdout, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stdout, "\n");
        fprintf(stdout, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stdout, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stdout, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

    if (deviceProp.major < 1)
    {
        fprintf(stdout, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);
    }
    
    checkCudaErrors( hipSetDevice(devID) );
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::Init()
{
	size_t vec_size = sigma.num_rows;
	VOID_vec.resize(vec_size, 0);
	NOT_FALSE_vec.resize(vec_size, 1);
	FALSE_vec.resize(vec_size, 0);
	BOOL_vec.resize(vec_size, 0);
	NUM_vec.resize(vec_size, 0); 
	LIST_vec.resize(vec_size, 0);
	tb.resize(1, 0);
	fb.resize(1, 0);
	AND_vec1.resize(r.size(), 0);
	AND_vec2.resize(r.size(), 0);

	for(int i=0; i<ARG_MAX; ++i)
	{
		v[i].resize(vec_size, 0);
		a[i].resize(sigma.num_cols, 0);
		
		if(Call[i].size() != r.size())
			Call[i].resize(r.size(), 0);
		if(PrimList[i].size() != r.size())
			PrimList[i].resize(r.size(), 0);
	}
#if(BUILD_TYPE == GPU)
	for(int i=0; i<NUM_STREAMS; ++i)
	{
		a_var[i].resize(sigma.num_cols, 0);
	    vf[i].resize(vec_size, 0);
	}
#else
	a_var.resize(sigma.num_cols, 0);
	vf.resize(vec_size, 0);
#endif
    a_set.resize(sigma.num_cols, 0);
    v_set.resize(vec_size, 0);
    v_cond.resize(vec_size, 0);
    v_list.resize(vec_size, 0);

	for(int i=0; i<vec_size; ++i)
	{
		if(i == vec_size - 5)			//list
			LIST_vec[i] = 1;			
		else if(i == vec_size - 4)		//void
			VOID_vec[i] = 1;
		else if(i == vec_size - 3)		//#t
			BOOL_vec[i] = 1;
		else if(i == vec_size - 2)		//#f
		{
			NOT_FALSE_vec[i] = 0;
			BOOL_vec[i] = 1;
			FALSE_vec[i] = 1;
		}
		else if(i == vec_size - 1)		//NUM
			NUM_vec[i] = 1;
	}

	#if(BUILD_TYPE == GPU)
    hipStreamCreate(&stream_Call);
    hipStreamCreate(&stream_List);
    hipStreamCreate(&stream_Set);
    hipStreamCreate(&stream_IF);
    hipStreamCreate(&stream_Num);
    hipStreamCreate(&stream_Bool);
    hipStreamCreate(&stream_Void);
    entry_count.resize(NUM_STREAMS, 0);

    //stream_count.resize(thrust::round_i(NUM_STREAMS, 32), 0);
	temp_indices.resize(r.size(), 0);
	temp_r.resize(r.size(), 0);
	for(int i=0; i<NUM_STREAMS; ++i)
	{
		s[i].resize(r.size(), 0);
		s_indices[i].resize(r.size(), 0);
		Fun_vec[i].resize(Fun.num_rows, 0);
		Body_vec[i].resize(Body.num_rows, 0);
		Arg_vec[i].resize(Arg[0].num_rows, 0);
		accum_var_vec[i].resize(sigma.num_cols);
		accum_vf_vec[i].resize(vec_size);
	}
	Cond_vec.resize(CondTrue.num_rows);
	for(int i=0; i<4*NUM_STREAMS; ++i)
		temp_Mat[i].resize(sigma.num_rows, sigma.num_cols, 0, std::max(sigma.num_cols/20, ulong(32)), 32);
	#else
	s.resize(r.size(), 0);
	s_indices.resize(r.size(), 0);
	#endif

	GRID = dim3(1,1,1);
	BLOCK = dim3(BLOCK_THREADS,1,1);
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::ReadTestFile(const char* filename)
{
	std::ifstream tf(filename);
	if(tf.fail())
	{
		fprintf(stdout, "Invalid test file: %s\n", filename);
		exit(1);
	}

	fprintf(stdout, "Reading test file: %s\n", filename);
	while(!tf.eof())
	{
		int rows, cols, i, j;
		char buf[64], name[32];

		tf.getline(buf, 64);
		if(tf.gcount() > 1)
		{
			std::sscanf(buf, "%s %d %d", name, &rows, &cols);

			std::vector< std::pair<int, int> > indices;
			tf.getline(buf, 64);
			while(tf.gcount() > 1)
			{
				std::sscanf(buf, "%d %d", &i, &j);
				if(i<0 || i>=rows || j<0 || j>=cols)
					fprintf(stdout, "ERROR Rows: %d Cols: %d \t (i:%d j:%d)\n", rows, cols, i, j);

				indices.push_back(std::make_pair(i,j));
				tf.getline(buf, 64);
			}
			std::sort(indices.begin(), indices.end(), compare_entry);

			#define NUM_ENTRIES_PER_ROW		32
			cusp::coo_matrix<int, int, cusp::host_memory> A(rows, cols, indices.size()), B;
			for(int i=0; i<A.num_entries; ++i)
			{
				A.row_indices[i] = indices[i].first;
				A.column_indices[i] = indices[i].second;
				A.values[i] = 1;
			}
			//A.sort_by_row_and_column();

			cusp::array1d<int, cusp::host_memory> vec;
			if(A.num_cols == 1)
			{
				vec.resize(A.num_rows, 0);
				for(int i=0; i<A.num_entries; ++i)
					vec[A.row_indices[i]] = 1;
			}

			fprintf(stderr, "\n%s (%d x %d)\n", name, A.num_rows, A.num_cols);
			B.resize(A.num_rows, A.num_cols, A.num_entries);
			cusp::transpose(A, B);
			fprintf(stderr, "B: (%d x %d)\n", B.num_rows, B.num_cols);
			cusp::print(B);

			std::string mat_name(name), sname = "", snum = "";
			int mat_num = -1;
			for(int i=0; i<NUM_MATRIX_TYPES; ++i)
			{
				std::string find_name = MatrixTypeMap[i];
				if(mat_name.find(find_name) == 0)
				{
					sname = find_name;
					snum = mat_name.substr(sname.size());
					if(snum.size() > 0)
						mat_num = atoi(snum.c_str());
					break;
				}
			}

			//parse name
			if(sname == "r")
				r = vec;
			else if(sname == "sigma")
				sigma = B;
			else if(sname == "CondTrue")
				CondTrue = B;
			else if(sname == "CondFalse")
				CondFalse = B;
			else if(sname == "Body")
				Body = B;
			else if(sname == "Fun")
				Fun = B;
			else if(sname == "Arg")
				Arg[mat_num-1] = B;
			else if(sname == "Var")
				Var[mat_num-1] = B;
			else if(sname == "Call")
			{
				Call[mat_num] = vec;
				if(m_maxCall < mat_num)
					m_maxCall = mat_num;
			}
			else if(sname == "PrimBool")
				PrimBool = vec;
			else if(sname == "PrimNum")
				PrimNum = vec;
			else if(sname == "PrimVoid")
				PrimVoid = vec;
			else if(sname == "PrimList")
			{
				PrimList[mat_num] = vec;
				if(m_maxList < mat_num)
					m_maxList = mat_num;
			}
			else if(sname == "If")
				IF = vec;
			else if(sname == "Set")
				SET = vec;
			else
				fprintf(stdout, "could not match input matrix: %s\n", name);
		}
	}

	tf.close();
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::Run_Analysis()
{
	fprintf(stdout, "\n\n\nStarting analysis...\n");
	r_prime = r;
	sigma_prime = sigma;
	int iter=0;
	bool sigma_change = false, r_change = false;
	fprintf(stderr, "m_maxCall: %d  m_maxList: %d\n", m_maxCall, m_maxList);
	cusp::ell_matrix<int, INDEX_TYPE, cusp::host_memory> temp_sigma;

	debug = false;
	//double time1=0, time2=0, time3=0, time4=0, time5=0, time6=0, time7=0;
	//double startTime=0, endTime=0;

	//#ifdef _OPENMP
	//#pragma omp parallel num_threads(1)
	do
	{
		int ID = omp_get_thread_num();
		if(ID == 0)
			fprintf(stdout, "\n\nITERATION %d\n\n", ++iter);

		if(iter == 10)
			debug = true;
		else
			debug = false;

#if BUILD_TYPE == GPU
		// //if(ID == 1)
		 	f_call();
		// //if(ID == 2)
		// 	f_list();
		// //if(ID == 3)
			f_set();
		// //if(ID == 4)
			f_if();
		// //if(ID == 5)
		 	f_primBool();
		// //if(ID == 6)
		 	f_primNum();
		//if(ID == 7)
			f_primVoid();
#else
		f_call();
		// 	f_list();
		f_set();
		f_if();
	 	f_primBool();
	 	f_primNum();
		f_primVoid();
#endif

	//#pragma omp barrier
		if(ID == 0)
		{
			fprintf(stdout, "\nupdate sigma\n");
		#if BUILD_TYPE == GPU
			//sigma_prime.num_entries = CountEntries(sigma_prime);


			// cusp::coo_matrix<INDEX_TYPE, VALUE_TYPE, MEM_TYPE> temp_sigma;
			// fprintf(stdout, "***sigma difference***\n");
			// cusp::subtract(sigma_prime, sigma, temp_sigma);
			// cusp::print(temp_sigma);
		#else
			for(int i=0; i<sigma_prime.num_entries; ++i)
				sigma_prime.values[i] = 1;

			// cusp::coo_matrix<INDEX_TYPE, VALUE_TYPE, MEM_TYPE> temp_sigma;
			// fprintf(stdout, "***sigma difference***\n");
			// cusp::subtract(sigma_prime, sigma, temp_sigma);
			// cusp::print(temp_sigma);
		#endif
			sigma_prime.num_entries = thrust::count_if(sigma_prime.column_indices.values.begin(), sigma_prime.column_indices.values.end(), is_non_negative());
			fprintf(stderr, "num_entries: %d  %d\n", sigma.num_entries, sigma_prime.num_entries);
			if(sigma.num_entries != sigma_prime.num_entries)
				sigma_change = true;
			else
				sigma_change = false;

			fprintf(stdout, "\nupdate r\n");
			int r_entries = thrust::count(r.begin(), r.end(), 1);
			int r_prime_entries = thrust::count(r_prime.begin(), r_prime.end(), 1);

			//thrust::transform(r.begin(), r.end(), r_prime.begin(), temp_r.begin(), thrust::minus<INDEX_TYPE>());
			// if(debug)
			// {
			// 	fprintf(stdout, "***r difference***\n");
			// 	cusp::print(temp_r);
			// 	cusp::print(r_prime);
			// }
			if(r_entries != r_prime_entries)
				r_change = true;
			else
				r_change = false;

			r = r_prime;
			sigma = sigma_prime;
			fprintf(stdout, "end iteration\n");
		}

	//#pragma omp barrier
	} while(r_change || sigma_change);
	//#endif

	fprintf(stdout, "Analysis Complete...\n");
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::WriteStore()
{
	// fprintf(stdout, "\nr:\n");
	// cusp::print(r);

#if BUILD_TYPE == GPU
	int num_entries = CountEntries(sigma);
	sigma.num_entries = num_entries;
#endif

	cusp::coo_matrix<int, VALUE_TYPE, cusp::host_memory> store;
	cusp::transpose(sigma, store);
	// fprintf(stdout, "\nsigma:\n");
	// cusp::print(store);

	std::ofstream output("tests/output.dat");
	output << "sigma " << store.num_rows << " " << store.num_cols << std::endl;
	for(int i=0; i<store.num_entries; ++i)
	{
		output << store.row_indices[i] << " " << store.column_indices[i] << " " << std::endl;
	}

	output.close();
}

void CPU_Test(std::string filename)
{
#if(BUILD_TYPE == CPU)
	CFA<int, int, cusp::host_memory> Analysis;
#elif(BUILD_TYPE == GPU)
	CFA<int, int, cusp::device_memory> Analysis;
	Analysis.gpuDeviceInit(0);
#endif	

	Analysis.ReadTestFile(filename.c_str());
	Analysis.Init();

	double startTime = omp_get_wtime();
	Analysis.Run_Analysis();
	double endTime = omp_get_wtime();

	fprintf(stdout, "Run Time: %f seconds\n", endTime - startTime);
	Analysis.WriteStore();
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
int CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::CountEntries(cusp::ell_matrix<INDEX_TYPE, VALUE_TYPE, cusp::device_memory> &mat)
{
	cusp::ell_matrix<INDEX_TYPE, VALUE_TYPE, cusp::host_memory> temp(mat);
	const INDEX_TYPE invalid_index = cusp::ell_matrix<int, INDEX_TYPE, cusp::device_memory>::invalid_index;

	int num_entries = 0;
	int pitch = temp.column_indices.pitch;
	for(int col=0; col<temp.column_indices.num_cols; ++col)
	{
		int offset = pitch*col;
		for(int row=0; row<temp.num_rows; ++row, ++offset)
		{
			if(temp.column_indices.values[offset] != invalid_index)
				num_entries++;
		}
	}
	temp.num_entries = num_entries;
	//cusp::print(temp);

	return num_entries;
}

void GPU_Test()
{

}