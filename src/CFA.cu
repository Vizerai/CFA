#include "CFA.cuh"
#include "CFA.h"

#define DEBUG		1

#if(DEBUG == 1)
#define MV_MULTIPLY(A, B, C)	cusp::multiply(A, B, C);\
								fprintf(stdout, "(%dx%d) * (%dx1) -> (%dx1)\n", A.num_rows, A.num_cols, B.size(), C.size())

#define DEBUG_PRINT(A, B)		fprintf(stdout, A);\
								cusp::print(B);
#else
#define MULTIPLY(A, B, C)		cusp::multiply(A, B, C)
#define DEBUG_PRINT(A, B)
#endif

#include "host_forms.inl"
#include "device_forms.inl"

// template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
// inline void subtract_vec(cusp::array1d<INDEX_TYPE, MEM_TYPE> &A,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &B,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &C)
// {
// 	thrust::transform(A.begin(), A.end(), B.begin(), C.begin(), thrust::minus<INDEX_TYPE>());
// }

// General GPU Device CUDA Initialization
template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
int CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::gpuDeviceInit(int devID)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stdout, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stdout, "\n");
        fprintf(stdout, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stdout, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stdout, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

    if (deviceProp.major < 1)
    {
        fprintf(stdout, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);
    }
    
    checkCudaErrors( hipSetDevice(devID) );
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::Init()
{
	size_t vec_size = sigma.num_rows;
	VOID_vec.resize(vec_size, 0);
	NOT_FALSE_vec.resize(vec_size, 1);
	FALSE_vec.resize(vec_size, 0);
	BOOL_vec.resize(vec_size, 0);
	NUM_vec.resize(vec_size, 0); 
	LIST_vec.resize(vec_size, 0);
	tb.resize(1, 0);
	fb.resize(1, 0);
	AND_vec1.resize(r.size(), 0);
	AND_vec2.resize(r.size(), 0);

	for(int i=0; i<ARG_MAX; ++i)
	{
		v[i].resize(vec_size, 0);
		a[i].resize(sigma.num_cols, 0);
		
		if(Call[i].size() != r.size())
			Call[i].resize(r.size(), 0);
		if(PrimList[i].size() != r.size())
			PrimList[i].resize(r.size(), 0);
	}
    a_var.resize(sigma.num_cols, 0);
    a_set.resize(sigma.num_cols, 0);
    vf.resize(vec_size, 0);
    v_set.resize(vec_size, 0);
    v_cond.resize(vec_size, 0);
    v_list.resize(vec_size, 0);

	for(int i=0; i<vec_size; ++i)
	{
		if(i == vec_size - 5)			//list
			LIST_vec[i] = 1;			
		else if(i == vec_size - 4)		//void
			VOID_vec[i] = 1;
		else if(i == vec_size - 3)		//#t
			BOOL_vec[i] = 1;
		else if(i == vec_size - 2)		//#f
		{
			NOT_FALSE_vec[i] = 0;
			BOOL_vec[i] = 1;
			FALSE_vec[i] = 1;
		}
		else if(i == vec_size - 1)		//NUM
			NUM_vec[i] = 1;
	}

	#if(BUILD_TYPE == GPU)
    hipStreamCreate(&stream_Call);
    hipStreamCreate(&stream_List);
    hipStreamCreate(&stream_Set);
    hipStreamCreate(&stream_IF);
    hipStreamCreate(&stream_Num);
    hipStreamCreate(&stream_Bool);
    hipStreamCreate(&stream_Void);

	s_indices.resize(r.size());
	temp_indices.resize(r.size());
	temp_r.resize(r.size());
	Fun_vec.resize(Fun.num_rows);
	Body_vec.resize(Body.num_rows);
	Arg_vec.resize(Arg[0].num_rows);
	Cond_vec.resize(CondTrue.num_rows);
	accum_var_vec.resize(sigma.num_cols);
	accum_vf_vec.resize(vec_size);
	temp_vec.resize(sigma.num_cols);
	temp_Mat[0].resize(sigma.num_rows, sigma.num_cols, 0, sigma.num_cols/20, 32);
	temp_Mat[1].resize(sigma.num_rows, sigma.num_cols, 0, sigma.num_cols/20, 32);
	temp_Mat[2].resize(sigma.num_rows, sigma.num_cols, 0, sigma.num_cols/20, 32);
	temp_Mat[3].resize(sigma.num_rows, sigma.num_cols, 0, sigma.num_cols/20, 32);
	temp_Mat[4].resize(sigma.num_rows, sigma.num_cols, 0, sigma.num_cols/20, 32);
	#endif

	GRID = dim3(1,1,1);
	BLOCK = dim3(BLOCK_THREADS,1,1);
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::ReadTestFile(const char* filename)
{
	std::ifstream tf(filename);
	if(tf.fail())
	{
		fprintf(stdout, "Invalid test file: %s\n", filename);
		exit(1);
	}

	fprintf(stdout, "Reading test file: %s\n", filename);
	while(!tf.eof())
	{
		int rows, cols, i, j;
		char buf[64], name[32];

		tf.getline(buf, 64);
		if(tf.gcount() > 1)
		{
			std::sscanf(buf, "%s %d %d", name, &rows, &cols);

			std::vector< std::pair<int, int> > indices;
			tf.getline(buf, 64);
			while(tf.gcount() > 1)
			{
				std::sscanf(buf, "%d %d", &i, &j);
				if(i<0 || i>=rows || j<0 || j>=cols)
					fprintf(stdout, "ERROR Rows: %d Cols: %d \t (i:%d j:%d)\n", rows, cols, i, j);

				indices.push_back(std::make_pair(i,j));
				tf.getline(buf, 64);
			}
			std::sort(indices.begin(), indices.end(), compare_entry);

			#define NUM_ENTRIES_PER_ROW		32
			cusp::coo_matrix<int, int, cusp::host_memory> A(rows, cols, indices.size()), B;
			for(int i=0; i<A.num_entries; ++i)
			{
				A.row_indices[i] = indices[i].first;
				A.column_indices[i] = indices[i].second;
				A.values[i] = 1;
			}
			//A.sort_by_row_and_column();

			cusp::array1d<int, cusp::host_memory> vec;
			if(A.num_cols == 1)
			{
				vec.resize(A.num_rows, 0);
				for(int i=0; i<A.num_entries; ++i)
					vec[A.row_indices[i]] = 1;
			}

			fprintf(stderr, "\n%s (%d x %d)\n", name, A.num_rows, A.num_cols);
			B.resize(A.num_rows, A.num_cols, A.num_entries);
			cusp::transpose(A, B);
			fprintf(stderr, "B: (%d x %d)\n", B.num_rows, B.num_cols);
			cusp::print(B);

			std::string mat_name(name), sname = "", snum = "";
			int mat_num = -1;
			for(int i=0; i<NUM_MATRIX_TYPES; ++i)
			{
				std::string find_name = MatrixTypeMap[i];
				if(mat_name.find(find_name) == 0)
				{
					sname = find_name;
					snum = mat_name.substr(sname.size());
					if(snum.size() > 0)
						mat_num = atoi(snum.c_str());
					break;
				}
			}

			//parse name
			if(sname == "r")
				r = vec;
			else if(sname == "sigma")
				sigma = B;
			else if(sname == "CondTrue")
				CondTrue = B;
			else if(sname == "CondFalse")
				CondFalse = B;
			else if(sname == "Body")
				Body = B;
			else if(sname == "Fun")
				Fun = B;
			else if(sname == "Arg")
				Arg[mat_num-1] = B;
			else if(sname == "Var")
				Var[mat_num-1] = B;
			else if(sname == "Call")
			{
				Call[mat_num] = vec;
				if(m_maxCall < mat_num)
					m_maxCall = mat_num;
			}
			else if(sname == "PrimBool")
				PrimBool = vec;
			else if(sname == "PrimNum")
				PrimNum = vec;
			else if(sname == "PrimVoid")
				PrimVoid = vec;
			else if(sname == "PrimList")
			{
				PrimList[mat_num] = vec;
				if(m_maxList < mat_num)
					m_maxList = mat_num;
			}
			else if(sname == "If")
				IF = vec;
			else if(sname == "Set")
				SET = vec;
			else
				fprintf(stdout, "could not match input matrix: %s\n", name);
		}
	}

	tf.close();
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::Run_Analysis()
{
	fprintf(stdout, "\n\n\nStarting analysis...\n");

	s.resize(r.size(), 0);
	r_prime = r;
	sigma_prime = sigma;
	int iter=0;
	bool sigma_change = false, r_change = false;
	fprintf(stderr, "m_maxCall: %d  m_maxList: %d\n", m_maxCall, m_maxList);
	cusp::ell_matrix<int, INDEX_TYPE, cusp::host_memory> temp_sigma;

	debug = false;
	double time1=0, time2=0, time3=0, time4=0, time5=0, time6=0, time7=0;
	double startTime=0, endTime=0;

	do
	{
		//CPU execution
		fprintf(stdout, "\n\nITERATION %d\n\n", ++iter);

		//startTime = omp_get_wtime();
		//f_call
		f_call();
		// endTime = omp_get_wtime();
		// time1 += endTime - startTime;
		// fprintf(stderr, "time Call: %f\n", time1);

		//startTime = omp_get_wtime();
		//f_list
		f_list();
		// endTime = omp_get_wtime();
		// time2 += endTime - startTime;
		// fprintf(stderr, "time List: %f\n", time2);

		//startTime = omp_get_wtime();
		//f_set
		f_set();
		// endTime = omp_get_wtime();
		// time3 += endTime - startTime;
		// fprintf(stderr, "time Set: %f\n", time3);

		//startTime = omp_get_wtime();
		//f_if
		f_if();
		// endTime = omp_get_wtime();
		// time4 += endTime - startTime;
		// fprintf(stderr, "time IF: %f\n", time4);

		//startTime = omp_get_wtime();
		//f_primBool
		f_primBool();
		// endTime = omp_get_wtime();
		// time5 += endTime - startTime;
		// fprintf(stderr, "time Bool: %f\n", time5);

		//startTime = omp_get_wtime();
		//f_primNum
		f_primNum();
		// endTime = omp_get_wtime();
		// time6 += endTime - startTime;
		// fprintf(stderr, "time Num: %f\n", time6);

		//startTime = omp_get_wtime();
		//f_primVoid
		f_primVoid();
		// endTime = omp_get_wtime();
		// time7 += endTime - startTime;
		// fprintf(stderr, "time Void: %f\n", time7);

		//correct values...  (this won't be a problem in later versions)
		// for(int i=0; i<sigma_prime.num_entries; ++i)
		// 	sigma_prime.values[i] = 1;
		// for(int i=0; i<r_prime.size(); ++i)
		// 	if(r_prime[i])
		// 		r_prime[i] = 1;

		fprintf(stdout, "\nupdate sigma\n");
		// cusp::subtract(sigma_prime, sigma, temp_Mat[0]);
		// if(debug)
		// {
		//  	fprintf(stdout, "***sigma difference***\n");
		// 	//cusp::print(temp_Mat[0]);

		// 	temp_sigma = sigma_prime;
		// 	int num_entries = 0;
		// 	int pitch = temp_sigma.column_indices.pitch;
		// 	const INDEX_TYPE invalid_index = cusp::ell_matrix<int, INDEX_TYPE, cusp::device_memory>::invalid_index;
		// 	for(int col=0; col<temp_sigma.column_indices.num_cols; ++col)
		// 	{	
		// 		int offset = pitch*col;
		// 		for(int row=0; row<temp_sigma.num_rows; ++row, ++offset)
		// 		{
		// 			if(temp_sigma.column_indices.values[offset] != invalid_index)
		// 				num_entries++;
		// 		}
		// 	}
		// 	temp_sigma.num_entries = num_entries;
		// 	cusp::print(temp_sigma);
		// }
		if(sigma.num_entries != sigma_prime.num_entries)
			sigma_change = true;
		else
			sigma_change = false;

		fprintf(stdout, "\nupdate r\n");
		int r_entries = thrust::count(r.begin(), r.end(), 1);
		int r_prime_entries = thrust::count(r_prime.begin(), r_prime.end(), 1);

		//thrust::transform(r.begin(), r.end(), r_prime.begin(), temp_r.begin(), thrust::minus<INDEX_TYPE>());
		// if(debug)
		// {
		// 	fprintf(stdout, "***r difference***\n");
		// 	cusp::print(temp_r);
		// 	cusp::print(r_prime);
		// }
		if(r_entries != r_prime_entries)
			r_change = true;
		else
			r_change = false;

		r = r_prime;
		sigma = sigma_prime;
		fprintf(stdout, "end iteration\n");

	} while(r_change || sigma_change || iter < 1030);

	fprintf(stdout, "Analysis Complete...\n");
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::WriteStore()
{
	// fprintf(stdout, "\nr:\n");
	// cusp::print(r);

	cusp::coo_matrix<int, VALUE_TYPE, MEM_TYPE> store;
	cusp::transpose(sigma, store);
	// fprintf(stdout, "\nsigma:\n");
	// cusp::print(store);

	std::ofstream output("tests/output.dat");
	output << "sigma " << store.num_rows << " " << store.num_cols << std::endl;
	for(int i=0; i<store.num_entries; ++i)
	{
		output << store.row_indices[i] << " " << store.column_indices[i] << " " << std::endl;
	}

	output.close();
}

void CPU_Test(std::string filename)
{
#if(BUILD_TYPE == CPU)
	CFA<int, int, cusp::host_memory> Analysis;
#elif(BUILD_TYPE == GPU)
	CFA<int, char, cusp::device_memory> Analysis;
	Analysis.gpuDeviceInit(0);
#endif	

	Analysis.ReadTestFile(filename.c_str());
	Analysis.Init();

	double startTime = omp_get_wtime();
	Analysis.Run_Analysis();
	double endTime = omp_get_wtime();

	fprintf(stdout, "Run Time: %f seconds\n", endTime - startTime);
	Analysis.WriteStore();
}

void GPU_Test()
{

}