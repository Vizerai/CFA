#include "hip/hip_runtime.h"
#include "CFA.cuh"
#include "CFA.h"

#define DEBUG		1

#if(DEBUG == 1)
#define MV_MULTIPLY(A, B, C)	cusp::multiply(A, B, C);\
								fprintf(stdout, "(%dx%d) * (%dx1) -> (%dx1)\n", A.num_rows, A.num_cols, B.size(), C.size())

#define DEBUG_PRINT(A, B)		fprintf(stdout, A);\
								cusp::print(B);
#else
#define MULTIPLY(A, B, C)		cusp::multiply(A, B, C)
#define DEBUG_PRINT(A, B)
#endif

#include "host_forms.inl"
#include "device_forms.inl"

// template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
// inline void subtract_vec(cusp::array1d<INDEX_TYPE, MEM_TYPE> &A,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &B,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &C)
// {
// 	thrust::transform(A.begin(), A.end(), B.begin(), C.begin(), thrust::minus<INDEX_TYPE>());
// }

// General GPU Device CUDA Initialization
// int gpuDeviceInit(int devID)
// {
//     int deviceCount;
//     checkCudaErrors(hipGetDeviceCount(&deviceCount));

//     if (deviceCount == 0)
//     {
//         fprintf(stdout, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
//         exit(-1);
//     }

//     if (devID < 0)
//        devID = 0;
        
//     if (devID > deviceCount-1)
//     {
//         fprintf(stdout, "\n");
//         fprintf(stdout, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
//         fprintf(stdout, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
//         fprintf(stdout, "\n");
//         return -devID;
//     }

//     hipDeviceProp_t deviceProp;
//     checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

//     if (deviceProp.major < 1)
//     {
//         fprintf(stdout, "gpuDeviceInit(): GPU device does not support CUDA.\n");
//         exit(-1);
//     }
    
//     checkCudaErrors( hipSetDevice(devID) );
//     checkCudaErrors( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );
//     printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

//     return devID;
// }

int * shared_store::host_column_indices_UA = NULL;
int * shared_store::host_values_UA = NULL;
int * shared_store::host_column_indices = NULL;
int * shared_store::host_values = NULL;
size_t shared_store::num_rows = 0;
size_t shared_store::num_cols = 0;
size_t shared_store::num_cols_per_row = 0;
size_t shared_store::pitch = 0;
size_t shared_store::num_entries = 0;

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::Init()
{
	size_t vec_size = sigma.num_rows;
	VOID_vec.resize(vec_size, 0);
	NOT_FALSE_vec.resize(vec_size, 1);
	FALSE_vec.resize(vec_size, 0);
	BOOL_vec.resize(vec_size, 0);
	NUM_vec.resize(vec_size, 0); 
	LIST_vec.resize(vec_size, 0);
	tb.resize(1, 0);
	fb.resize(1, 0);
	AND_vec1.resize(r.size(), 0);
	AND_vec2.resize(r.size(), 0);

	for(int i=0; i<ARG_MAX; ++i)
	{
		v[i].resize(vec_size, 0);
		a[i].resize(sigma.num_cols, 0);
		
		if(Call[i].size() != r.size())
			Call[i].resize(r.size(), 0);
		if(PrimList[i].size() != r.size())
			PrimList[i].resize(r.size(), 0);
	}
#if(BUILD_TYPE == GPU)
	for(int i=0; i<NUM_STREAMS; ++i)
	{
		a_var[i].resize(sigma.num_cols, 0);
	    vf[i].resize(vec_size, 0);
	}
#else
	a_var.resize(sigma.num_cols, 0);
	vf.resize(vec_size, 0);
#endif
    a_set.resize(sigma.num_cols, 0);
    v_set.resize(vec_size, 0);
    v_cond.resize(vec_size, 0);
    v_list.resize(vec_size, 0);

	for(int i=0; i<vec_size; ++i)
	{
		if(i == vec_size - 5)			//list
			LIST_vec[i] = 1;			
		else if(i == vec_size - 4)		//void
			VOID_vec[i] = 1;
		else if(i == vec_size - 3)		//#t
			BOOL_vec[i] = 1;
		else if(i == vec_size - 2)		//#f
		{
			NOT_FALSE_vec[i] = 0;
			BOOL_vec[i] = 1;
			FALSE_vec[i] = 1;
		}
		else if(i == vec_size - 1)		//NUM
			NUM_vec[i] = 1;
	}

	#if(BUILD_TYPE == GPU)
    hipStreamCreate(&stream_Call);
    hipStreamCreate(&stream_List);
    hipStreamCreate(&stream_Set);
    hipStreamCreate(&stream_IF);
    hipStreamCreate(&stream_Num);
    hipStreamCreate(&stream_Bool);
    hipStreamCreate(&stream_Void);

	a_indices.resize(sigma.num_cols, 0);
	v_indices.resize(vec_size, 0);
	for(int i=0; i<NUM_STREAMS; ++i)
	{
		index_count[i].resize(8, 0);
		temp_row_indices[i].resize(vec_size, 0);
		temp_col_indices[i].resize(sigma.num_cols, 0);
		s[i].resize(r.size(), 0);
		s_indices[i].resize(r.size(), 0);
		Fun_vec[i].resize(Fun.num_rows, 0);
		Body_vec[i].resize(Body.num_rows, 0);
		Arg_vec[i].resize(Arg[0].num_rows, 0);
		accum_var_vec[i].resize(sigma.num_cols);
		accum_vf_vec[i].resize(vec_size);
	}
	Cond_vec.resize(CondTrue.num_rows);
	for(int i=0; i<4*NUM_STREAMS; ++i)
		temp_Mat[i].resize(sigma.num_rows, sigma.num_cols, 0, std::max(sigma.num_cols/20, ulong(32)), 32);
	#else
	s.resize(r.size(), 0);
	s_indices.resize(r.size(), 0);
	#endif
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::ReadTestFile(const char* filename)
{
	int ID = omp_get_thread_num();

	std::ifstream tf(filename);
	if(tf.fail())
	{
		fprintf(stdout, "Invalid test file: %s\n", filename);
		exit(1);
	}

	fprintf(stdout, "Reading test file: %s\n", filename);
	while(!tf.eof())
	{
		int rows, cols, i, j;
		char buf[64], name[32];
		valid_Call.resize(ARG_MAX);
		valid_List.resize(ARG_MAX);

		tf.getline(buf, 64);
		if(tf.gcount() > 1)
		{
			std::sscanf(buf, "%s %d %d", name, &rows, &cols);

			std::vector< std::pair<int, int> > indices;
			tf.getline(buf, 64);
			while(tf.gcount() > 1)
			{
				std::sscanf(buf, "%d %d", &i, &j);
				if(i<0 || i>=rows || j<0 || j>=cols)
					fprintf(stdout, "ERROR Rows: %d Cols: %d \t (i:%d j:%d)\n", rows, cols, i, j);

				indices.push_back(std::make_pair(i,j));
				tf.getline(buf, 64);
			}
			std::sort(indices.begin(), indices.end(), compare_entry);

			//check name
			std::string mat_name(name), sname = "", snum = "";
			int mat_num = -1;
			for(int i=0; i<NUM_MATRIX_TYPES; ++i)
			{
				std::string find_name = MatrixTypeMap[i];
				if(mat_name.find(find_name) == 0)
				{
					sname = find_name;
					snum = mat_name.substr(sname.size());
					if(snum.size() > 0)
						mat_num = atoi(snum.c_str());
					break;
				}
			}

			cusp::coo_matrix<int, int, cusp::host_memory> A(rows, cols, indices.size());
			cusp::coo_matrix<int, int, cusp::host_memory> B(cols, rows, indices.size());

			for(int i=0; i<indices.size(); ++i)
			{
				A.row_indices[i] = indices[i].first;
				A.column_indices[i] = indices[i].second;
				A.values[i] = 1;
			}
			A.sort_by_row_and_column();
			cusp::transpose(A, B);
			B.sort_by_row_and_column();

			cusp::array1d<int, cusp::host_memory> vec;
			if(A.num_cols == 1)
			{
				vec.resize(A.num_rows, 0);
				for(int i=0; i<A.num_entries; ++i)
				{
					vec[A.row_indices[i]] = 1;
				}
			}

			if(ID == 0 && A.num_entries > 0)
			{
				fprintf(stderr, "\n%s (%d x %d) with %d entries\n", name, A.num_rows, A.num_cols, A.num_entries);
				fprintf(stderr, "B: (%d x %d)\n", B.num_rows, B.num_cols);
			}

			//parse name
			if(sname == "r")
				r = vec;
			else if(sname == "sigma")
			{
#if BUILD_TYPE == CPU
				sigma = B;
#else			
				if(ID == 0)
				{
					shared_sigma.num_rows = B.num_rows;
					shared_sigma.num_cols = B.num_cols;
					shared_sigma.num_cols_per_row = std::max(B.num_cols/20, ulong(32));
					shared_sigma.pitch = B.num_rows;
					shared_sigma.num_entries = B.num_entries;

					size_t index_size = shared_sigma.pitch  * shared_sigma.num_cols_per_row * sizeof(INDEX_TYPE);
					size_t values_size = shared_sigma.pitch * shared_sigma.num_cols_per_row * sizeof(VALUE_TYPE);

					// checkCudaErrors(hipHostAlloc((void **)&shared_sigma.host_column_indices, index_size, hipHostMallocMapped));
					// checkCudaErrors(hipHostAlloc((void **)&shared_sigma.host_values, values_size, hipHostMallocMapped));

					size_t entry_count_size = 32*sizeof(INDEX_TYPE);
					checkCudaErrors( hipHostAlloc((void **)&entry_count_host, entry_count_size, 0));
					checkCudaErrors( hipMalloc((void **)&entry_count_device, entry_count_size));
					memset(entry_count_host, 0, entry_count_size);
					checkCudaErrors( hipMemcpy(entry_count_device, entry_count_host, entry_count_size, hipMemcpyHostToDevice) );
				}
				#pragma omp barrier
				//checkCudaErrors(hipHostGetDevicePointer((void **)&shared_sigma.column_indices, (void *)shared_sigma.host_column_indices, 0));
				//checkCudaErrors(hipHostGetDevicePointer((void **)&shared_sigma.values, (void *)shared_sigma.host_values, 0));
				//checkCudaErrors(hipHostGetDevicePointer((void **)&entry_count_device, (void *)entry_count_host, 0));
				#pragma omp barrier
				cusp::csr_matrix<INDEX_TYPE, VALUE_TYPE, cusp::device_memory> temp;
				temp = B;
				LoadEllMatrix(temp, sigma);
				int num_entries = thrust::count_if(sigma.column_indices.values.begin(), sigma.column_indices.values.end(), is_non_negative());
				fprintf(stderr, "num_entries: %d\n", num_entries);

				// LoadEllMatrix(temp, shared_sigma);
				// fprintf(stderr, "shared_sigma.num_cols: %d\n", shared_sigma.num_cols);
				// fprintf(stderr, "shared_sigma.num_rows: %d\n", shared_sigma.num_rows);
#endif
				if(ID == 0)
					print_matrix_info(sigma);
			}
			else if(sname == "CondTrue")
			{
				CondTrue = B;
				if(ID == 0)
					print_matrix_info(CondTrue);
			}
			else if(sname == "CondFalse")
			{
				CondFalse = B;
				if(ID == 0)
					print_matrix_info(CondFalse);
			}
			else if(sname == "Body")
			{
				Body = B;
				if(ID == 0)
					print_matrix_info(Body);
			}
			else if(sname == "Fun")
			{
				Fun = B;
				if(ID == 0)
					print_matrix_info(Fun);
			}
			else if(sname == "Arg")
			{
				Arg[mat_num-1] = B;
				if(ID == 0)
					print_matrix_info(Arg[mat_num-1]);
			}
			else if(sname == "Var")
			{
				Var[mat_num-1] = B;
				if(ID == 0)
					print_matrix_info(Var[mat_num-1]);
			}
			else if(sname == "Call")
			{
				if(A.num_entries > 0)
					valid_Call[mat_num] = true;
				else
					valid_Call[mat_num] = false;

				Call[mat_num] = vec;
				if(m_maxCall < mat_num)
					m_maxCall = mat_num;
			}
			else if(sname == "PrimBool")
				PrimBool = vec;
			else if(sname == "PrimNum")
				PrimNum = vec;
			else if(sname == "PrimVoid")
				PrimVoid = vec;
			else if(sname == "PrimList")
			{
				if(A.num_entries > 0)
					valid_List[mat_num] = true;
				else
					valid_List[mat_num] = false;

				PrimList[mat_num] = vec;
				if(m_maxList < mat_num)
					m_maxList = mat_num;
			}
			else if(sname == "If")
				IF = vec;
			else if(sname == "Set")
				SET = vec;
			else
				fprintf(stdout, "could not match input matrix: %s\n", name);
		}
	}

	tf.close();
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::Run_Analysis()
{
	int ID = omp_get_thread_num();
	fprintf(stdout, "\n\n\nStarting analysis: %d\n", ID);

	int prev_num_entries = sigma.num_entries;
	r_prime = r;
	int iter=0;
	bool sigma_change = false, r_change = false;
	fprintf(stderr, "m_maxCall: %d  m_maxList: %d\n", m_maxCall, m_maxList);

	//#pragma omp parallel num_threads(NUM_STREAMS)
	do
	{
		iter++;
		if(ID == 0)
			fprintf(stdout, "\n\nITERATION %d\n\n", iter);

		// if(iter == 12)
		// 	debug = true;
		// else
		// 	debug = false;

#if BUILD_TYPE == GPU
		//if(ID == 1)
		 	f_call();
		//if(ID == 2)
			f_list();
		//if(ID == 3)
			f_set();
		//if(ID == 4)
			f_if();
		//if(ID == 5)
		 	f_primBool();
		//if(ID == 6)
			f_primNum();
		//if(ID == 7)
			f_primVoid();
#else
		f_call();
		f_list();
		f_set();
		f_if();
	 	f_primBool();
	 	f_primNum();
		f_primVoid();
#endif

	//#pragma omp barrier
		if(ID == 0 && iter % 5 == 0)
		{
			fprintf(stdout, "\nupdate sigma\n");
		#if BUILD_TYPE == GPU
			// #if MULTI_GPU == 2
			// int count = 0;
			// for(int row=0; row<shared_sigma.num_rows; ++row)
			// {
			// 	int offset = row;
			// 	for(int col=0; col < shared_sigma.num_cols_per_row; ++col, offset+=shared_sigma.pitch)
			// 	{
			// 		if(shared_sigma.column_indices[offset] != -1)
			// 			count++;
			// 	}
			// }
			// shared_sigma.num_entries = count;
			// #else
			if(iter % 5 == 0)
			sigma.num_entries = thrust::count_if(sigma.column_indices.values.begin(), sigma.column_indices.values.end(), is_non_negative());
			fprintf(stderr, "num_entries: %d  %d\n", prev_num_entries, sigma.num_entries);
			//DEBUG_PRINT("sigma", sigma);

			//#endif
		#else
			//sigma.num_entries = thrust::count_if(sigma.column_indices.values.begin(), sigma.column_indices.values.end(), is_non_negative());
			//thrust::fill(sigma.values.begin(), sigma.values.end(), 1);
		#endif

			if(prev_num_entries != sigma.num_entries)
				sigma_change = true;
			else
				sigma_change = false;
			prev_num_entries = sigma.num_entries;

			fprintf(stdout, "\nupdate r\n");
			int r_entries = thrust::count(r.begin(), r.end(), 1);
			int r_prime_entries = thrust::count(r_prime.begin(), r_prime.end(), 1);

			if(r_entries != r_prime_entries)
				r_change = true;
			else
				r_change = false;

			r = r_prime;
			fprintf(stderr, "sigma.num_entries: %d\n", sigma.num_entries);
			//sigma = sigma_prime;
			fprintf(stdout, "end iteration\n");
		}
		else
		{
			r_change = true;
			sigma_change = true;
		}

	//#pragma omp barrier
	} while(r_change || sigma_change);

	fprintf(stdout, "Analysis Complete...\n");
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::WriteStore()
{
	// fprintf(stdout, "\nr:\n");
	// cusp::print(r);

#if BUILD_TYPE == GPU
	int num_entries = CountEntries(sigma);
	sigma.num_entries = num_entries;
#endif

	cusp::coo_matrix<int, VALUE_TYPE, cusp::host_memory> store;
	cusp::transpose(sigma, store);
	// fprintf(stdout, "\nsigma:\n");
	// cusp::print(store);

	std::ofstream output("tests/output.dat");
	output << "sigma " << store.num_rows << " " << store.num_cols << std::endl;
	for(int i=0; i<store.num_entries; ++i)
	{
		output << store.row_indices[i] << " " << store.column_indices[i] << " " << std::endl;
	}

	output.close();
}

void Test(std::string filename)
{
	double startTime, endTime;
#if BUILD_TYPE == CPU
	CFA<int, int, cusp::host_memory> Analysis;

	Analysis.ReadTestFile(filename.c_str());
	Analysis.Init();

	startTime = omp_get_wtime();
	Analysis.Run_Analysis();
	endTime = omp_get_wtime();

	fprintf(stdout, "Run Time: %f seconds\n", endTime - startTime);
	Analysis.WriteStore();

#elif BUILD_TYPE == GPU
	#pragma omp parallel num_threads(NUM_GPUS)
	{
		int ID = omp_get_thread_num();
		gpuDeviceInit(ID);
		fprintf(stderr, "thread ID: %d\n", ID);
		CFA<int, int, cusp::device_memory> Analysis;

		Analysis.ReadTestFile(filename.c_str());
		Analysis.Init();

	#pragma omp barrier

		if(ID == 0)
			startTime = omp_get_wtime();
		
		Analysis.Run_Analysis();

		if(ID == 0)
		{
			endTime = omp_get_wtime();
			fprintf(stdout, "Run Time: %f seconds\n", endTime - startTime);
			Analysis.WriteStore();
		}
	}
#endif

	
}

template <typename INDEX_TYPE, typename VALUE_TYPE, typename MEM_TYPE>
int CFA<INDEX_TYPE, VALUE_TYPE, MEM_TYPE>::CountEntries(cusp::ell_matrix<INDEX_TYPE, VALUE_TYPE, cusp::device_memory> &mat)
{
	cusp::ell_matrix<INDEX_TYPE, VALUE_TYPE, cusp::host_memory> temp(mat);
	const INDEX_TYPE invalid_index = cusp::ell_matrix<int, INDEX_TYPE, cusp::device_memory>::invalid_index;

	int num_entries = 0;
	int pitch = temp.column_indices.pitch;
	for(int col=0; col<temp.column_indices.num_cols; ++col)
	{
		int offset = pitch*col;
		for(int row=0; row<temp.num_rows; ++row, ++offset)
		{
			if(temp.column_indices.values[offset] != invalid_index)
				num_entries++;
		}
	}
	temp.num_entries = num_entries;
	//cusp::print(temp);

	return num_entries;
}