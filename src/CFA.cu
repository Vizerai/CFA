#include "hip/hip_runtime.h"
#include "CFA.cuh"
#include "CFA.h"

#define BLOCK_SIZE	256
#define DEBUG	1

#if(DEBUG == 1)
#define MULTIPLY(A, B, C)		cusp::multiply(A, B, C);\ 
								fprintf(stderr, "(%d,%d) x (%d,%d) -> (%d,%d)\n", A.num_rows, A.num_cols, B.num_rows, B.num_cols, C.num_rows, C.num_cols)

#define DEBUG_PRINT(A, B)		fprintf(stderr, A);\
								cusp::print(B);
#else
#define MULTIPLY(A, B, C)		cusp::multiply(A, B, C)
#define DEBUG_PRINT(A, B)
#endif

#include "host_forms.inl"
#include "device_forms.inl"

// template <typename INDEX_TYPE, typename MEM_TYPE>
// inline void subtract_vec(cusp::array1d<INDEX_TYPE, MEM_TYPE> &A,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &B,
// 							cusp::array1d<INDEX_TYPE, MEM_TYPE> &C)
// {
// 	thrust::transform(A.begin(), A.end(), B.begin(), C.begin(), thrust::minus<INDEX_TYPE>());
// }

// General GPU Device CUDA Initialization
template <typename INDEX_TYPE, typename MEM_TYPE>
int CFA<INDEX_TYPE, MEM_TYPE>::gpuDeviceInit(int devID)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stderr, "\n");
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stderr, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

    if (deviceProp.major < 1)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);
    }
    
    checkCudaErrors( hipSetDevice(devID) );
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::AND_OP(	const cusp::array1d<INDEX_TYPE, MEM_TYPE> &A, 
										const cusp::array1d<INDEX_TYPE, MEM_TYPE> &B, 
										cusp::array1d<INDEX_TYPE, MEM_TYPE> &C)
{
	assert(A.size() == B.size());
	C.resize(A.size(), 0);

	for(int i=0; i<A.size(); ++i)
		C[i] = A[i] & B[i];
}

// template <typename IndexType, typename MemorySpace>
// __launch_bounds__(BLOCK_SIZE,1)
// __global__ void CFA<INDEX_TYPE, MEM_TYPE>::AND_OP(	const IndexType *A, 
// 										const IndexType *B, 
// 										IndexType *C,
// 										const IndexType size)
// {
// 	const int threadID = blockIdx.x*blockDim.x + threadIdx.x;
// 	const int grid_size = blockDim.x*gridDim.x;

// 	for(int i=threadID; i<size; i+=grid_size)
// 		C[i] = A[i] & B[i];
// }

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::Init()
{
	tb.resize(1, 0);
	fb.resize(1, 0);
	VOID_vec.resize(sigma.num_cols, 0);
	NOT_FALSE_vec.resize(sigma.num_cols, 1);
	FALSE_vec.resize(sigma.num_cols, 0);
	BOOL_vec.resize(sigma.num_cols, 0);
	NUM_vec.resize(sigma.num_cols, 0);

	int max = sigma.num_cols;
	for(int i=0; i<max; ++i)
	{
		if(i == max - 4)			//void
			VOID_vec[i] = 1;
		else if(i == max - 3)		//#t
			BOOL_vec[i] = 1;
		else if(i == max - 2)		//#f
		{
			NOT_FALSE_vec[i] = 0;
			BOOL_vec[i] = 1;
			FALSE_vec[i] = 1;
		}
		else if(i == max - 1)		//NUM
			NUM_vec[i] = 1;
	}

	m_ValuesSize = sigma.num_rows;
}

// template <typename INDEX_TYPE, typename MEM_TYPE>
// void CFA<INDEX_TYPE, MEM_TYPE>::Vectorize(	cusp::ell_matrix<int,char,cusp::host_memory> &A, 
// 												cusp::ell_matrix<int,char,cusp::host_memory> &B)
// {
// 	B.resize(1, A.num_rows*A.num_cols, A.num_entries);
// 	for(int i=0; i<A.column_indices.size(); ++i)
// 	{
// 		B.row_indices[i] = 0;
// 		B.column_indices[i] = A.row_indices[i]*A.num_cols + A.column_indices[i];
// 		B.values[i] = 1;
// 	}
// 	B.sort_by_row_and_column();
// }

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::ReadTestFile(const char* filename)
{
	std::ifstream tf(filename);
	if(tf.fail())
	{
		fprintf(stderr, "Invalid test file: %s\n", filename);
		exit(1);
	}

	fprintf(stderr, "Reading test file: %s\n", filename);
	while(!tf.eof())
	{
		int rows, cols, i, j;
		char buf[64], name[32];

		tf.getline(buf, 64);
		if(tf.gcount() > 1)
		{
			std::sscanf(buf, "%s %d %d", name, &rows, &cols);
			//std::cout << name << "  " << rows << "  " << cols << std::endl;

			std::vector< std::pair<int, int> > indices;
			tf.getline(buf, 64);
			while(tf.gcount() > 1)
			{
				std::sscanf(buf, "%d %d", &i, &j);
				indices.push_back(std::make_pair(i,j));
				//fprintf(stderr, "gcout: %d %d %d\n", tf.gcount(), x, y);
				tf.getline(buf, 64);
			}
			std::sort(indices.begin(), indices.end());

			#define NUM_ENTRIES_PER_ROW		32
			cusp::coob_matrix<int,cusp::host_memory> A(rows, cols, indices.size()), B;
			for(int i=0; i<indices.size(); ++i)
			{
				A.row_indices[i] = indices[i].first;
				A.column_indices[i] = indices[i].second;
			}
			A.sort_by_row_and_column();

			cusp::array1d<INDEX_TYPE, MEM_TYPE> vec;
			if(A.num_cols == 1)
			{
				vec.resize(A.num_rows, 0);
				for(int i=0; i<A.num_entries; ++i)
					vec[A.row_indices[i]] = 1;
			}

			fprintf(stderr, "\n%s (%d x %d)\n", name, A.num_rows, A.num_cols);
			cusp::transpose(A, B);
			//cusp::print(A);

			std::string mat_name(name), sname = "", snum = "";
			int mat_num = -1;
			for(int i=0; i<NUM_MATRIX_TYPES; ++i)
			{
				std::string find_name = MatrixTypeMap[i];
				if(mat_name.find(find_name) == 0)
				{
					sname = find_name;
					snum = mat_name.substr(sname.size());
					if(snum.size() > 0)
						mat_num = atoi(snum.c_str()) - 1;
					break;
				}
			}

			//fprintf(stderr, "mat_name: %s \t\t sname: %s\n", mat_name.c_str(), sname.c_str());
			//parse name
			if(sname == "r")
				r = vec;
			else if(sname == "sigma")
				sigma = A;
			else if(sname == "CondTrue")
				CondTrue = A;
			else if(sname == "CondFalse")
				CondFalse = A;
			else if(sname == "Body")
				Body = A;
			else if(sname == "Fun")
				Fun = A;
			else if(sname == "Arg")
				Arg[mat_num] = A;
			else if(sname == "Var")
				Var[mat_num] = A;
			else if(sname == "Call")
			{
				Call[mat_num] = B;
				if(m_maxArgs < mat_num + 1)
					m_maxArgs = mat_num + 1;
			}
			else if(sname == "PrimBool")
				PrimBool = vec;
			else if(sname == "PrimNum")
				PrimNum = vec;
			else if(sname == "PrimVoid")
				PrimVoid = vec;
			else if(sname == "If")
				IF = vec;
			else if(sname == "Set")
				SET = vec;
			else
				fprintf(stderr, "could not match input matrix: %s\n", name);
		}
	}

	tf.close();
}

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::Run_Analysis()
{
	fprintf(stderr, "\n\n\nStarting analysis...\n");

	//kernel calls
	cusp::array1d<INDEX_TYPE, MEM_TYPE> s(sigma.num_cols, 0);
	fprintf(stderr, "m_maxArgs: %d\n", m_maxArgs);
	bool sigma_change = false, r_change = false;
	r_prime = r;
	sigma_prime = sigma;
	int iter=0;

	do
	{
		//CPU execution
		fprintf(stderr, "\n\nITERATION %d\n\n", ++iter);
		//f_call
		// fprintf(stderr, "f_call\n");
	 // 		for(int j=1; j<=m_maxArgs; ++j)
		// {
		// 	//r & Call
		// 	AND_OP(r, Call[j-1], s);
		// 	f_call(s, j);
		// }

		//f_set
		fprintf(stderr, "f_set\n");
		AND_OP(r, SET, s);
		f_set(s);

		//f_if
		// fprintf(stderr, "f_if\n");
		// AND_OP(r, IF, s);
		// f_if(s);

		// //f_primBool
		// fprintf(stderr, "f_primBool\n");
		// AND_OP(r, PrimBool, s);
		// f_primBool(s);

		// //f_primNum
		// fprintf(stderr, "f_primNum\n");
		// AND_OP(r, PrimInt, s);
		// f_primNum(s);

		// //f_primVoid
		// fprintf(stderr, "f_PrimVoid\n");
		// AND_OP(r, PrimVoid, s);
		// f_primVoid(s);

		//correct values...  (this won't be a problem in later versions)
		// for(int i=0; i<sigma_prime.num_entries; ++i)
		// 	sigma_prime.values[i] = 1;
		// for(int i=0; i<r_prime.num_entries; ++i)
		// 	r_prime.values[i] = 1;

		// fprintf(stderr, "\n\nupdate sigma\n");
		// cusp::subtract(sigma, sigma_prime, temp_Mat[0]);
		// fprintf(stderr, "***sigma difference***\n");
		// cusp::print(temp_Mat[0]);
		// sigma_change = false;
		// for(int i=0; i<temp_Mat[0].num_entries; ++i)
		// 	if(temp_Mat[0].values[i] != 0)
		// 	{
		// 		sigma_change = true;
		// 		break;
		// 	}

		fprintf(stderr, "\n\nupdate r\n");
		//subtract_vec(r, r_prime, temp_vec);
		thrust::transform(r.begin(), r.end(), r_prime.begin(), temp_vec.begin(), thrust::minus<INDEX_TYPE>());
		// fprintf(stderr, "***r difference***\n");
		// cusp::print(temp_vec);
		r_change = false;
		for(int i=0; i<temp_vec.size(); ++i)
			if(temp_vec[i] != 0)
			{
				r_change = true;
				break;
			}

		sigma_prime.sort_by_row_and_column();
		r = r_prime;
		sigma = sigma_prime;

//#elif(MEM_TYPE == DEVICE)
		//GPU kernel invocation

		//dim3 GRID, BLOCK;

		//GPU_Iterate<<<GRID, BLOCK>>> ();
		//read back change vectors

//#endif
	} while(r_change || sigma_change);

	fprintf(stderr, "Analysis Complete...\n");
}

template <typename INDEX_TYPE, typename MEM_TYPE>
void CFA<INDEX_TYPE, MEM_TYPE>::WriteStore()
{
	fprintf(stderr, "\nsigma:\n");
	cusp::print(sigma);
	fprintf(stderr, "\nr:\n");
	cusp::print(r);
	std::ofstream output("tests/output.dat");

	output << "sigma " << sigma.num_cols << " " << sigma.num_rows << std::endl;
	for(int i=0; i<sigma.num_entries; ++i)
	{
		output << sigma.row_indices[i] << " " << sigma.column_indices[i] << " " << std::endl;
	}

	output.close();
}

void CPU_Test(std::string filename)
{
	CFA<char, cusp::host_memory> Analysis;
	//Analysis.gpuDeviceInit(1);

	Analysis.ReadTestFile(filename.c_str());
	Analysis.Init();
	//Analysis.PrintMatrices();

	// double startTime = omp_get_wtime();
	// Analysis.Run_Analysis();
	// double endTime = omp_get_wtime();

	// fprintf(stderr, "Run Time: %f seconds\n", endTime - startTime);
	// Analysis.WriteStore();
}

void GPU_Test()
{

}